#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <vector>
#include <fstream>
#include <hip/hip_runtime.h>
#include <map>
#include <hip/hip_runtime.h>
#include <>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

using namespace std;

//gpu stuff

#define NoPOLY 256
//#define MAXCCPOLY 200         //gpu block number

//Timing stuff

#define TIMESTEPS 10000
#define SUBDIFFUSE 5000         //21, do y=100000 41, do y=200000 61, do y=500000 81, do y=750000 121, do y=2000000 101, do y=1600000
#define DIFFUSE 7500           //81, do y=1250000 101, do y=2000000

//length of polymer

#define POLYLENGTH 21
#define RESOLUTION 1                  //segment length //carefull drastically reduces number of conformations
#define NANOSIZE 0              //size of nanoparticle
#define DENSITY 2               //NANOSIZE:DENSITY makes the ratio, with 1, 2, 3 .......DENSITY    where NANOSIZE fills 1 if 1,  1,2 if 2 1,2,3 if 3 etc etc.


class Managed
{
public:
  void *operator new(size_t len) {
    void *ptr;
    hipMallocManaged(&ptr, len);
    return ptr;
  }

  void operator delete(void *ptr) {
    hipFree(ptr);
  }
};
	

class statistics{
      
private:
int n;
double sum;
double sumsq;

public:
statistics();
int getNumber() const;
double getAverage() const;
double getSqAverage() const;
void add(double x);
};
	
struct Polymer : public Managed{
              
			  int Gridx[POLYLENGTH];
			  int Gridy[POLYLENGTH];
			  int Gridz[POLYLENGTH];
			  int Randoz[POLYLENGTH];
			  int Randoz2[POLYLENGTH];
			  //float Stats[150000];
			  int tracker;
			  int check;
              int currentnode;
              int upnode;
              int downnode;
              int xsame;
              int ysame;
              int zsame;
              int randomdir;
              double endtoend;
              double beadtomid;
              double radofgy;
              int block;
              int resloop;
              int nloopx;
              int nloopy;
              int nloopz;
              double smidx;
              double smidy;
              double smidz;
			  void intial();
			  void polylength();
			  void com();
			  void comin();
			  void gyration();
			  void random();
             };
	
	
__global__ void cudarandomwalk(Polymer *polymer) {

	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < NoPOLY)	
	{
		//polymer[idx].check++;
	    //printf("checking threading on device = %d  \n", POLYLENGTH);
		//printf("checking threading on device = %d  \n", polymer[idx].check);
		for (int z=0; z<(POLYLENGTH*polymer[idx].check); z++)
		{
			polymer[idx].xsame = 0; //random
			polymer[idx].ysame = 0;
			polymer[idx].zsame = 0;
			polymer[idx].block = 0;

			polymer[idx].currentnode = polymer[idx].Randoz[z];
			polymer[idx].randomdir =  polymer[idx].Randoz2[z];
			//polymer[idx].currentnode = 1;
			//polymer[idx].randomdir =  1;

			polymer[idx].upnode=(polymer[idx].currentnode+1);
			polymer[idx].downnode=(polymer[idx].currentnode-1);

			if ((0 < polymer[idx].currentnode) && (polymer[idx].currentnode < (POLYLENGTH-1)))
			{
				if (polymer[idx].Gridx[polymer[idx].downnode] == polymer[idx].Gridx[polymer[idx].upnode])   polymer[idx].xsame = 1;
				if (polymer[idx].Gridy[polymer[idx].downnode] == polymer[idx].Gridy[polymer[idx].upnode])   polymer[idx].ysame = 1;
				if (polymer[idx].Gridz[polymer[idx].downnode] == polymer[idx].Gridz[polymer[idx].upnode])   polymer[idx].zsame = 1;
			}

			if (polymer[idx].currentnode == 0)
			{
				polymer[idx].Gridx[polymer[idx].currentnode] = polymer[idx].Gridx[polymer[idx].upnode];
				polymer[idx].Gridy[polymer[idx].currentnode] = polymer[idx].Gridy[polymer[idx].upnode];
				polymer[idx].Gridz[polymer[idx].currentnode] = polymer[idx].Gridz[polymer[idx].upnode];
				if (polymer[idx].randomdir == 0) polymer[idx].Gridx[polymer[idx].currentnode]--;          
				else if (polymer[idx].randomdir == 1) polymer[idx].Gridx[polymer[idx].currentnode]++;        
				else if (polymer[idx].randomdir == 2) polymer[idx].Gridy[polymer[idx].currentnode]--;
				else if (polymer[idx].randomdir == 3) polymer[idx].Gridy[polymer[idx].currentnode]++; 
				else if (polymer[idx].randomdir == 4) polymer[idx].Gridz[polymer[idx].currentnode]--;   
				else polymer[idx].Gridz[polymer[idx].currentnode]++;    
			}

			if (polymer[idx].currentnode == (POLYLENGTH-1))
			{
				polymer[idx].Gridx[polymer[idx].currentnode] = polymer[idx].Gridx[polymer[idx].downnode];
				polymer[idx].Gridy[polymer[idx].currentnode] = polymer[idx].Gridy[polymer[idx].downnode];
				polymer[idx].Gridz[polymer[idx].currentnode] = polymer[idx].Gridz[polymer[idx].downnode];
				if (polymer[idx].randomdir == 0) polymer[idx].Gridx[polymer[idx].currentnode]--;          
				else if (polymer[idx].randomdir == 1) polymer[idx].Gridx[polymer[idx].currentnode]++;        
				else if (polymer[idx].randomdir == 2) polymer[idx].Gridy[polymer[idx].currentnode]--;
				else if (polymer[idx].randomdir == 3) polymer[idx].Gridy[polymer[idx].currentnode]++; 
				else if (polymer[idx].randomdir == 4) polymer[idx].Gridz[polymer[idx].currentnode]--;   
				else polymer[idx].Gridz[polymer[idx].currentnode]++;    
			}

			if ((0 < polymer[idx].currentnode) && (polymer[idx].currentnode < (POLYLENGTH-1)))
			{
				if ((polymer[idx].xsame == 1) && (polymer[idx].ysame == 1) && (polymer[idx].zsame == 1))
				{
				polymer[idx].Gridx[polymer[idx].currentnode] = polymer[idx].Gridx[polymer[idx].upnode];
				polymer[idx].Gridy[polymer[idx].currentnode] = polymer[idx].Gridy[polymer[idx].upnode];
				polymer[idx].Gridz[polymer[idx].currentnode] = polymer[idx].Gridz[polymer[idx].upnode];
				if (polymer[idx].randomdir == 0) polymer[idx].Gridx[polymer[idx].currentnode]--;          
				else if (polymer[idx].randomdir == 1) polymer[idx].Gridx[polymer[idx].currentnode]++;        
				else if (polymer[idx].randomdir == 2) polymer[idx].Gridy[polymer[idx].currentnode]--;
				else if (polymer[idx].randomdir == 3) polymer[idx].Gridy[polymer[idx].currentnode]++; 
				else if (polymer[idx].randomdir == 4) polymer[idx].Gridz[polymer[idx].currentnode]--;   
				else polymer[idx].Gridz[polymer[idx].currentnode]++;    
				}
			}
		}
    //polymer[idx].currentnode++;
	//polymer[idx].randomdir++;
    }
}

__global__ void cudarandomwalk2(Polymer *polymers) {

	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < NoPOLY)	
	{
		Polymer& polymer = polymers[idx];
		//polymer.check++;
	    //printf("checking threading on device = %d  \n", POLYLENGTH);
		//printf("checking threading on device = %d  \n", polymer.check);
		for (int z=0; z<(POLYLENGTH*polymer.check); z++)
		{
			polymer.xsame = 0; //random
			polymer.ysame = 0;
			polymer.zsame = 0;
			polymer.block = 0;

			polymer.currentnode = polymer.Randoz[z];
			polymer.randomdir =  polymer.Randoz2[z];
			//if (idx==0) printf("checking currentnode on device = %d  \n", polymer.currentnode);
		    //if (idx==0) printf("checking randomdir on device = %d  \n", polymer.randomdir);
			//polymer.currentnode = 1;
			//polymer.randomdir =  1;

			polymer.upnode=(polymer.currentnode+1);
			polymer.downnode=(polymer.currentnode-1);

			if ((0 < polymer.currentnode) && (polymer.currentnode < (POLYLENGTH-1)))
			{
				if (polymer.Gridx[polymer.downnode] == polymer.Gridx[polymer.upnode])   polymer.xsame = 1;
				if (polymer.Gridy[polymer.downnode] == polymer.Gridy[polymer.upnode])   polymer.ysame = 1;
				if (polymer.Gridz[polymer.downnode] == polymer.Gridz[polymer.upnode])   polymer.zsame = 1;
			}

			if (polymer.currentnode == 0)
			{
				polymer.Gridx[polymer.currentnode] = polymer.Gridx[polymer.upnode];
				polymer.Gridy[polymer.currentnode] = polymer.Gridy[polymer.upnode];
				polymer.Gridz[polymer.currentnode] = polymer.Gridz[polymer.upnode];
				if (polymer.randomdir == 0) polymer.Gridx[polymer.currentnode]--;          
				else if (polymer.randomdir == 1) polymer.Gridx[polymer.currentnode]++;        
				else if (polymer.randomdir == 2) polymer.Gridy[polymer.currentnode]--;
				else if (polymer.randomdir == 3) polymer.Gridy[polymer.currentnode]++; 
				else if (polymer.randomdir == 4) polymer.Gridz[polymer.currentnode]--;   
				else polymer.Gridz[polymer.currentnode]++;    
			}

			if (polymer.currentnode == (POLYLENGTH-1))
			{
				polymer.Gridx[polymer.currentnode] = polymer.Gridx[polymer.downnode];
				polymer.Gridy[polymer.currentnode] = polymer.Gridy[polymer.downnode];
				polymer.Gridz[polymer.currentnode] = polymer.Gridz[polymer.downnode];
				if (polymer.randomdir == 0) polymer.Gridx[polymer.currentnode]--;          
				else if (polymer.randomdir == 1) polymer.Gridx[polymer.currentnode]++;        
				else if (polymer.randomdir == 2) polymer.Gridy[polymer.currentnode]--;
				else if (polymer.randomdir == 3) polymer.Gridy[polymer.currentnode]++; 
				else if (polymer.randomdir == 4) polymer.Gridz[polymer.currentnode]--;   
				else polymer.Gridz[polymer.currentnode]++;    
			}

			if ((0 < polymer.currentnode) && (polymer.currentnode < (POLYLENGTH-1)))
			{
				if ((polymer.xsame == 1) && (polymer.ysame == 1) && (polymer.zsame == 1))
				{
				polymer.Gridx[polymer.currentnode] = polymer.Gridx[polymer.upnode];
				polymer.Gridy[polymer.currentnode] = polymer.Gridy[polymer.upnode];
				polymer.Gridz[polymer.currentnode] = polymer.Gridz[polymer.upnode];
				if (polymer.randomdir == 0) polymer.Gridx[polymer.currentnode]--;          
				else if (polymer.randomdir == 1) polymer.Gridx[polymer.currentnode]++;        
				else if (polymer.randomdir == 2) polymer.Gridy[polymer.currentnode]--;
				else if (polymer.randomdir == 3) polymer.Gridy[polymer.currentnode]++; 
				else if (polymer.randomdir == 4) polymer.Gridz[polymer.currentnode]--;   
				else polymer.Gridz[polymer.currentnode]++;    
				}
			}
		}
    //polymer.currentnode++;
	//polymer.randomdir++;
    }
}

void Polymer::intial()
{
       int forcedir = 3;
       int iran = 0;
       int i;
       int blocked = 0;
       int resloup = 0;
       int nloupx = 0;
       int nloupy = 0;
       int nloupz = 0;

       for(i=1; i < POLYLENGTH; i++)
       {
                  iran = rand()%forcedir;
                  blocked = 0;

                  if(iran==2)
                  {
                           for (resloup = 1; resloup <= RESOLUTION; resloup++)
                           {
                                    for (nloupx = 0; nloupx < NANOSIZE; nloupx++)
                                    {
                                               for (nloupy = 0; nloupy < NANOSIZE; nloupy++)
                                               {
                                                        for (nloupz = 0; nloupz < NANOSIZE; nloupz++)
                                                        {
                                                                if (((((Gridx[POLYLENGTH-1]+resloup)%DENSITY) == nloupx)) && ((((Gridy[POLYLENGTH-1])%DENSITY) == nloupy)) && ((((Gridz[POLYLENGTH-1])%DENSITY) == nloupz))) blocked = 1;
                                                        }
                                               }
                                    }
                           }
                           if (blocked == 0) Gridx[POLYLENGTH-1]++;
                  }

                  if(iran==1)
                  {
                           for (resloup = 1; resloup <= RESOLUTION; resloup++)
                           {
                                    for (nloupx = 0; nloupx < NANOSIZE; nloupx++)
                                    {
                                               for (nloupy = 0; nloupy < NANOSIZE; nloupy++)
                                               {
                                                        for (nloupz = 0; nloupz < NANOSIZE; nloupz++)
                                                        {
                                                                if (((((Gridx[POLYLENGTH-1])%DENSITY) == nloupx)) && ((((Gridy[POLYLENGTH-1]+resloup)%DENSITY) == nloupy)) && ((((Gridz[POLYLENGTH-1])%DENSITY) == nloupz))) blocked = 1;
                                                        }
                                               }
                                    }
                           }
                           if (blocked == 0) Gridy[POLYLENGTH-1]++;
                  }

                  if(iran==0)
                  {
                           for (resloup = 1; resloup <= RESOLUTION; resloup++)
                           {
                                    for (nloupx = 0; nloupx < NANOSIZE; nloupx++)
                                    {
                                               for (nloupy = 0; nloupy < NANOSIZE; nloupy++)
                                               {
                                                        for (nloupz = 0; nloupz < NANOSIZE; nloupz++)
                                                        {
                                                                if (((((Gridx[POLYLENGTH-1])%DENSITY) == nloupx)) && ((((Gridy[POLYLENGTH-1])%DENSITY) == nloupy)) && ((((Gridz[POLYLENGTH-1]+resloup)%DENSITY) == nloupz))) blocked = 1;
                                                        }
                                               }
                                    }
                           }
                           if (blocked == 0) Gridz[POLYLENGTH-1]++;
                  }

                  if (blocked == 0) 
					  {
						  Gridx[i]=Gridx[POLYLENGTH-1];
						  Gridy[i]=Gridy[POLYLENGTH-1];
						  Gridz[i]=Gridz[POLYLENGTH-1];
				      }
                  if (blocked == 1) i--;

       }
}

void Polymer::random()
{

int x;

	   for (x=0; x < (POLYLENGTH); x++)
	   {
	              Randoz[x]=rand()%POLYLENGTH;
				  Randoz2[x]=rand()%6;
	   }

}

void Polymer::polylength()
{
     double a;
     double b;
     double c;
     a = (Gridx[POLYLENGTH-1] - Gridx[0])*(Gridx[POLYLENGTH-1] - Gridx[0]);
     b = (Gridy[POLYLENGTH-1] - Gridy[0])*(Gridy[POLYLENGTH-1] - Gridy[0]);
     c = (Gridz[POLYLENGTH-1] - Gridz[0])*(Gridz[POLYLENGTH-1] - Gridz[0]);
     endtoend = sqrt(a+b+c);
}

void Polymer::comin()

{
     double x,y,z;
     int i=0;
     x = 0;
     y = 0;
     z = 0;

     for (i;i<POLYLENGTH;i++)
     {
         x =  x +(double(Gridx[i]));
         y =  y +(double(Gridy[i]));
         z =  z +(double(Gridz[i]));
     }

     smidx = x;
     smidy = y;
     smidz = z;
}


void Polymer::com()

{
     double x,y,z;
     int i=0;
     x = 0;
     y = 0;
     z = 0;


     for (i;i<POLYLENGTH;i++)
     {
         x =  x +(double(Gridx[i]));
         y =  y +(double(Gridy[i]));
         z =  z +(double(Gridz[i]));
     }

     x = (x - smidx)/double(POLYLENGTH);
     y = (y - smidy)/double(POLYLENGTH);
     z = (z - smidz)/double(POLYLENGTH);

     beadtomid = ((x*x)+(y*y)+(z*z));
}


void Polymer::gyration()

{
     double x,y,z,radx,rady,radz,vrad;
     int i=0;
     int j=0;
     x = 0;
     y = 0;
     z = 0;
     radx = 0;
     rady = 0;
     radz = 0;
     vrad = 0;

     for (i;i<POLYLENGTH;i++)
     {
         x =  x +(double(Gridx[i]));
         y =  y +(double(Gridy[i]));
         z =  z +(double(Gridz[i]));
     }

     x = (x)/(double(POLYLENGTH));
     y = (y)/(double(POLYLENGTH));
     z = (z)/(double(POLYLENGTH));

     for (j;j<POLYLENGTH;j++)
     {
     radx = (((double(Gridx[j])) - x)*((double(Gridx[j])) - x));
     rady = (((double(Gridy[j])) - y)*((double(Gridy[j])) - y));
     radz = (((double(Gridz[j])) - z)*((double(Gridz[j])) - z));
     vrad = vrad + radx + rady + radz;
     }

     radofgy = sqrt(vrad/(double(POLYLENGTH)));
}

statistics::statistics() {
n=0;
sum=0.0;
sumsq=0.0;
}

int statistics::getNumber() const{
return n;
}

double statistics::getAverage() const {
if(n==0) return 0.;
return sum/n;
}

double statistics::getSqAverage() const {
if(n==0) return -1;
return sumsq/n;
}

void statistics::add(double x) {
n++;
sum += x;
sumsq += x*x;
}

//-----------------------------------------------------------MAIN PROGRAM------------------------------------------------------------------------------------//

int main()
{
    long startTime = clock();
	long starttime2;
    long finishtime2;
	int i;
	int y;
	int x;
	//int k;
	int q;
	int g;

	srand(time(NULL));

	ofstream outfile;
    outfile.open ("TEST7.txt");//**************************************************************************************************************
    if (!outfile.is_open())
    { 
	    cout << "file not open" << endl;
		return 666;
	}

    outfile << "TimeStep " << "E2EDistance " << "R^2 " << "log10(TimeStep) " << "log10(R^2) " << endl;

	cout << "_____________" << endl << "STARTING STATS ..." << endl;

	//std::map<int, statistics> rsqmap;
	//std::map<int, statistics> flengthmap;

	//for (y=1; y<TIMESTEPS; y++)
	//{
	//	if ((y % 100) == 0 )
	//	{
	//		statistics rsq;
	//		statistics flength;
	//		rsqmap.insert(rsqmap.begin(), std::pair<int,statistics>(y, rsq));
	//	    flengthmap.insert(flengthmap.begin(), std::pair<int,statistics>(y, flength));
	//	}
	//}

		int polycount = NoPOLY;
		Polymer *Allpoly; 
		hipMallocManaged(&Allpoly, polycount * sizeof(Polymer));
	
		    for (i=0; i < polycount; i++) 
			{
				Allpoly[i].currentnode = 0;
				Allpoly[i].randomdir = 0;
				Allpoly[i].xsame = 0;
				Allpoly[i].ysame = 0;
				Allpoly[i].zsame = 0;
				Allpoly[i].endtoend = 0;
				Allpoly[i].radofgy = 0;
				Allpoly[i].beadtomid = 0;
				Allpoly[i].smidx = 0;
				Allpoly[i].smidy = 0;
				Allpoly[i].smidz = 0;
				Allpoly[i].tracker = 0;
				Allpoly[i].check = 1;
				for (q=0; q < POLYLENGTH; q++)
				{
					Allpoly[i].Gridx[q] = NANOSIZE;
					Allpoly[i].Gridy[q] = NANOSIZE;
					Allpoly[i].Gridz[q] = NANOSIZE;
			    }
				//for (q=0; q < POLYLENGTH; q++)
				//{
				//	Allpoly[i].Stats[q] = (float)q;
			 //   }
				Allpoly[i].intial();
				Allpoly[i].random();
			    Allpoly[i].comin();
            }
			Allpoly[0].polylength();
			cout << "_____________" << endl << "e2edistace=" << Allpoly[0].endtoend << endl << "_____________" << endl;
			for (g=0; g<POLYLENGTH; g++)
			{
				cout << "_____________________________" << endl;
				cout << "Bead no " << (g) <<" at (" << Allpoly[0].Gridx[g] << "," << Allpoly[0].Gridy[g] << "," << Allpoly[0].Gridz[g] << ")"  << endl;
			}

//-----------------------------------------------------------KERNAL CALL------------------------------------------------------------------------------------//

		    for (y=0; y<=TIMESTEPS; y++)
			{
				statistics rsq;
                statistics flength;
			    if ((y == 1000))  starttime2=clock();
				cudarandomwalk2<<<(polycount/(256))+1, polycount>>>(Allpoly);  //(polycount/(256))+1
                //cudarandomwalk<<<(polycount+255)/256, 256>>>(*Allpoly);
				hipDeviceSynchronize();
				if ((y == 1000))  finishtime2=clock();
	            if ((y == 1000)) cout<<endl<<"1 random walk takes "<<((finishtime2 - starttime2)/double(CLOCKS_PER_SEC))<<" seconds"<<endl<<endl;

				if ((y % 5000) == 0) cout << y << endl;
	    
				for (x=0; x < polycount; x++) 
				{
					Allpoly[x].random();
					
					if (y==SUBDIFFUSE)
					{
					     Allpoly[x].comin();
					}

					if ((y % 100) == 0 )
					{
						Allpoly[x].polylength();
						Allpoly[x].com();
						//rsqmap[y].add(Allpoly[x].beadtomid);
						//flengthmap[y].add(Allpoly[x].endtoend);
                        rsq.add(Allpoly[x].beadtomid);
                        flength.add(Allpoly[x].endtoend);

					}
				}
				if ((y % 100) == 0 ) 
                {
                                outfile << y << " " <<  flength.getAverage()  << " "  << rsq.getAverage() << " " << log10((double)(y)) << " " << log10(rsq.getAverage()) << endl;
                }
			}

			cout << "_____________" << endl << "END OF RANDOMWALK" << endl;
			
			Allpoly[0].polylength();
			cout << "_____________" << endl << "e2edistace=" << Allpoly[0].endtoend << endl << "_____________" << endl;
			
			for (g=0; g<POLYLENGTH; g++)
			{
				cout << "_____________________________" << endl;
				cout << "Bead no " << (g) <<" at (" << Allpoly[0].Gridx[g] << "," << Allpoly[0].Gridy[g] << "," << Allpoly[0].Gridz[g] << ")"  << endl;
			}
           
			hipFree(Allpoly);
	
	cout << "_____________" << endl << "PRINTING STATS ..." << endl;

	//for (y=1; y<TIMESTEPS; y++) //we can improve later
	//{
	//	if ((y % 100) == 0 )
	//	{
 //           statistics rsq = rsqmap[y];  
 //           statistics flength = flengthmap[y];  
	//		outfile << (y) << " " <<  flength.getAverage()  << " "  << rsq.getAverage() << " " << log10((double)(y)) << " " << log10(rsq.getAverage()) << endl;
	//	}
	//}

	cout << "_____________" << endl << "FINISHED STATS ..." << endl;
	
	hipDeviceReset();
	outfile.close();
	long finishTime = clock();
	cout<<endl<<"Run time is "<<((finishTime - startTime)/double(CLOCKS_PER_SEC))<<" seconds"<<endl<<endl;
	system("PAUSE");
    return 0;
}




